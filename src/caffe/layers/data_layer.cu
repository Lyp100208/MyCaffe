// Copyright 2013 Yangqing Jia

#include <stdint.h>
#include <leveldb/db.h>
#include <pthread.h>

#include <string>
#include <iostream>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/vision_layers.hpp"

using std::string;
using std::cout;

namespace caffe {

template <typename Dtype>
void DataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  //LOG(INFO)<<"(*top)[0]->num() = "<<(*top)[0]->num();
  // First, join the thread
  CHECK(!pthread_join(thread_, NULL)) << "Pthread joining failed.";
  // Copy the data
  CUDA_CHECK(hipMemcpy((*top)[0]->mutable_gpu_data(),
      prefetch_data_->cpu_data(), sizeof(Dtype) * prefetch_data_->count(),
      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy((*top)[1]->mutable_gpu_data(),
      prefetch_label_->cpu_data(), sizeof(Dtype) * prefetch_label_->count(),
      hipMemcpyHostToDevice));
  // Start a new prefetch thread
  CHECK(!pthread_create(&thread_, NULL, DataLayerPrefetch<Dtype>,
      reinterpret_cast<void*>(this))) << "Pthread execution failed.";
}
template <typename Dtype>
void DataLayer<Dtype>::MyForward_gpu(
      const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top, 
      const Dtype* data, const int len_data, 
      const Dtype* label, const int len_label) {
  //LOG(INFO)<<"MyForward_gpu";
  CUDA_CHECK(
    hipMemcpy(
      (*top)[0]->mutable_gpu_data(), data, sizeof(Dtype) * len_data, hipMemcpyHostToDevice
      )
    );
  CUDA_CHECK(
    hipMemcpy(
      (*top)[1]->mutable_gpu_data(), label, sizeof(Dtype) * len_label, hipMemcpyHostToDevice
      )
    );
}
// The backward operations are dummy - they do not carry any computation.
template <typename Dtype>
Dtype DataLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  return Dtype(0.);
}

INSTANTIATE_CLASS(DataLayer);

}  // namespace caffe
